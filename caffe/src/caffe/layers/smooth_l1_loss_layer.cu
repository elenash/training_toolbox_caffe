#include "hip/hip_runtime.h"
/*
All modification made by Intel Corporation: © 2016 Intel Corporation

All contributions by the University of California:
Copyright (c) 2014, 2015, The Regents of the University of California (Regents)
All rights reserved.

All other contributions:
Copyright (c) 2014, 2015, the respective contributors
All rights reserved.
For the list of contributors go to https://github.com/BVLC/caffe/blob/master/CONTRIBUTORS.md


Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are met:

    * Redistributions of source code must retain the above copyright notice,
      this list of conditions and the following disclaimer.
    * Redistributions in binary form must reproduce the above copyright
      notice, this list of conditions and the following disclaimer in the
      documentation and/or other materials provided with the distribution.
    * Neither the name of Intel Corporation nor the names of its contributors
      may be used to endorse or promote products derived from this software
      without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE
FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include <vector>

#include "caffe/layers/smooth_l1_loss_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void SmoothL1Forward(const int n, const Dtype* in, Dtype* out,
    Dtype sigma2) {
  // f(x) = 0.5 * (sigma * x)^2          if |x| < 1 / sigma / sigma
  //        |x| - 0.5 / sigma / sigma    otherwise
  CUDA_KERNEL_LOOP(index, n) {
    Dtype val = in[index];
    Dtype abs_val = abs(val);
    if (abs_val < 1.0 / sigma2) {
      out[index] = 0.5 * val * val * sigma2;
    } else {
      out[index] = abs_val - 0.5 / sigma2;
    }
  }
}

template <typename Dtype>
void SmoothL1LossLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  int count = bottom[0]->count();
  caffe_gpu_sub(
      count,
      bottom[0]->gpu_data(),
      bottom[1]->gpu_data(),
      diff_.mutable_gpu_data());    // d := b0 - b1
  if (has_weights_) {
    // apply "inside" weights
    caffe_gpu_mul(
        count,
        bottom[2]->gpu_data(),
        diff_.gpu_data(),
        diff_.mutable_gpu_data());  // d := w_in * (b0 - b1)
  }
  // NOLINT_NEXT_LINE(whitespace/operators)
  SmoothL1Forward<Dtype><<<CAFFE_GET_BLOCKS(count),
    CAFFE_CUDA_NUM_THREADS>>>(
      count, diff_.gpu_data(), errors_.mutable_gpu_data(), sigma2_);
  CUDA_POST_KERNEL_CHECK;

  if (has_weights_) {
    // apply "outside" weights
    caffe_gpu_mul(
        count,
        bottom[3]->gpu_data(),
        errors_.gpu_data(),
        errors_.mutable_gpu_data());  // d := w_out * SmoothL1(w_in * (b0 - b1))
  }

  Dtype loss;
  caffe_gpu_dot(count, ones_.gpu_data(), errors_.gpu_data(), &loss);
  top[0]->mutable_cpu_data()[0] = loss / bottom[0]->num();
}

template <typename Dtype>
__global__ void SmoothL1Backward(const int n, const Dtype* in, Dtype* out,
    Dtype sigma2) {
  // f'(x) = sigma * sigma * x         if |x| < 1 / sigma / sigma
  //       = sign(x)                   otherwise
  CUDA_KERNEL_LOOP(index, n) {
    Dtype val = in[index];
    Dtype abs_val = abs(val);
    if (abs_val < 1.0 / sigma2) {
      out[index] = sigma2 * val;
    } else {
      out[index] = (Dtype(0) < val) - (val < Dtype(0));
    }
  }
}

template <typename Dtype>
void SmoothL1LossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  // after forwards, diff_ holds w_in * (b0 - b1)
  int count = diff_.count();
  // NOLINT_NEXT_LINE(whitespace/operators)
  SmoothL1Backward<Dtype><<<CAFFE_GET_BLOCKS(count),
    CAFFE_CUDA_NUM_THREADS >>>(
      count, diff_.gpu_data(), diff_.mutable_gpu_data(), sigma2_);
  CUDA_POST_KERNEL_CHECK;
  for (int i = 0; i < 2; ++i) {
    if (propagate_down[i]) {
      const Dtype sign = (i == 0) ? 1 : -1;
      const Dtype alpha = sign * top[0]->cpu_diff()[0] / bottom[i]->num();
      caffe_gpu_axpby(
          count,                           // count
          alpha,                           // alpha
          diff_.gpu_data(),                // x
          Dtype(0),                        // beta
          bottom[i]->mutable_gpu_diff());  // y
      if (has_weights_) {
        // Scale by "inside" weight
        caffe_gpu_mul(
            count,
            bottom[2]->gpu_data(),
            bottom[i]->gpu_diff(),
            bottom[i]->mutable_gpu_diff());
        // Scale by "outside" weight
        caffe_gpu_mul(
            count,
            bottom[3]->gpu_data(),
            bottom[i]->gpu_diff(),
            bottom[i]->mutable_gpu_diff());
      }
    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(SmoothL1LossLayer);

}  // namespace caffe
